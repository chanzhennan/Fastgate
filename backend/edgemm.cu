#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>

#include "edgemm.cuh"

void edgemm(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 128, BN = 256, BK = 32;
    dim3 blockDim(256);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    const int NSPLIT = 4096;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    hipFuncSetAttribute(reinterpret_cast<const void*>(myHGEMMAlignedV5),
                hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    unsigned int dsmem = 2 * (BM * (BK + 8) + BK * (BN + 8)) * sizeof(half);
    
    myHGEMMAlignedV5<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}


void edgemm_m8n256k64(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 256, BK = 64;
    dim3 blockDim(256);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    const int NSPLIT = 4096;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemm_m8n256k64_v3),   
                hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    // about 76KB for m8n256k64
    // unsigned int dsmem = 2 * (BM * (8 * BK + 8) + BK * (BN + 8)) * sizeof(half);
    unsigned int dsmem = 2 * (BM * (4 * BK + 8) + BK * (BN + 8)) * sizeof(half);
    
    eed_hgemm_m8n256k64_v3<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}


void edgemm_m8n128k64(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 128, BK = 64;
    dim3 blockDim(128);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    const int NSPLIT = 2048;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    // hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemm_m8n256k64_v3),   
    //             hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    // about 39 KB for m8n128k64
    unsigned int dsmem = 2 * (BM * (2 * BK + 8) + BK * (BN + 8)) * sizeof(half);
    
    eed_hgemm_m8n128k64_v4<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}

void edgemm_m8n128k64x4(at::Tensor A, at::Tensor B, at::Tensor C) {
    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    if (N % 128 || K % 128) {
        throw std::invalid_argument("K(input column) & N(output column) must be multiple of 128!");
    }

    const int BM = 8, BN = 128, BK = 64;
    dim3 blockDim(128);
    int BX = N / BN;
    int BY = (M + BM - 1) / BM;
    int BZ = 1;

    int tile_num = BX * BY;
    if (tile_num <= 64) {
        BZ = 4;
        if (tile_num <= 32) {
            BZ = 8;
        }
    }

    if (K % 1024) {
        BZ = std::min(4, BZ);
        if (K % 512) {
            BZ = std::min(2, BZ);
            if (K % 256) {
                BZ = std::min(1, BZ);
            }
        }
    }

    half *output_ptr = reinterpret_cast<half *>(C.data_ptr<at::Half>());
    if (BZ > 1) {
        // invalid param
        // hipMemset(reinterpret_cast<void *>(C.data_ptr<at::Half>()), 0, K * N * sizeof(half));
        C.zero_();
    }

    dim3 gridDim(BX, BY, BZ);

    // about 36.25 KB
    uint smem_a = BM * (BK * 2 + 8);
    uint smem_b = 2 * BK * (BN + 8);
    unsigned int dsmem = (smem_a + smem_b) * sizeof(half);

    eed_hgemm_m8n128k64x4_v7<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),
        reinterpret_cast<half *>(B.data_ptr<at::Half>()),
        output_ptr,
        M, N, K);
}

// matric B(weight) transposed
void edgemm_m8n128k64x4_bt(at::Tensor A, at::Tensor B, at::Tensor C) {
    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(0);  // weight shape: N * K

    if (N % 128 || K % 128) {
        throw std::invalid_argument("K(input column) & N(transposed output row) must be multiple of 128!");
    }

    const int BM = 8, BN = 128, BK = 64;
    dim3 blockDim(128);
    int BX = N / BN;
    int BY = (M + BM - 1) / BM;
    int BZ = 1;

    int tile_num = BX * BY;
    if (tile_num <= 64) {
        BZ = 4;
        if (tile_num <= 32) {
            BZ = 8;
        }
    }

    if (K % 1024) {
        BZ = std::min(4, BZ);
        if (K % 512) {
            BZ = std::min(2, BZ);
            if (K % 256) {
                BZ = std::min(1, BZ);
            }
        }
    }

    half *output_ptr = reinterpret_cast<half *>(C.data_ptr<at::Half>());
    if (BZ > 1) {
        C.zero_();
    }

    dim3 gridDim(BX, BY, BZ);

    // about 38.125 KB
    uint smem_a = BM * (BK * 2 + 8);
    uint smem_b = 2 * BN * (BK + 8);
    unsigned int dsmem = (smem_a + smem_b) * sizeof(half);

    eed_hgemm_m8n128k64x4_v7_bt<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),
        reinterpret_cast<half *>(B.data_ptr<at::Half>()),
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),
        M, N, K);
}

void edgemm_m8n128k128(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 128, BK = 128;
    dim3 blockDim(128);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    const int NSPLIT = 2048;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemm_m8n128k128_v5),   
                hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    // about 74 KB for m8n128k128
    unsigned int dsmem = 2 * (BM * (BK + 8) + BK * (BN + 8)) * sizeof(half);
    
    eed_hgemm_m8n128k128_v5<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}


void edgemv_m1n128k64x4(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 128, BK = 64;
    dim3 blockDim(128);
    int BX = (N + BN - 1) / BN;
    int BY = 1;
    int BZ = 8;

    dim3 gridDim(BX, BY, BZ);


    // hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemv_m1n128k64_v6),   
    //             hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    // about 39 KB for m8n128k64
    unsigned int dsmem = (2 * (BM * (2 * BK + 8) + BK * (BN + 8)) + 0) * sizeof(half);
    
    eed_hgemv_m1n128k64x4_v6<8><<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}


void edgemv_m1n256k64x4(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 256, BK = 64;
    dim3 blockDim(256);
    int BX = (N + BN - 1) / BN;
    int BY = 1;
    int BZ = 8;

    dim3 gridDim(BX, BY, BZ);


    hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemv_m1n256k64x4_v8<8>),   
                hipFuncAttributeMaxDynamicSharedMemorySize, 76032);

    // about 76 KB for m8n256k64
    unsigned int dsmem = (2 * (BM * (4 * BK + 8) + BK * (BN + 8)) + 0) * sizeof(half);
    
    eed_hgemv_m1n256k64x4_v8<8><<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}
