#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>

#include "edgemm.cuh"

void edgemm(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 128, BN = 256, BK = 32;
    dim3 blockDim(256);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    const int NSPLIT = 4096;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    hipFuncSetAttribute(reinterpret_cast<const void*>(myHGEMMAlignedV5),
                hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    unsigned int dsmem = 2 * (BM * (BK + 8) + BK * (BN + 8)) * sizeof(half);
    
    myHGEMMAlignedV5<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}


void edgemm_m8n256k64(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 256, BK = 64;
    dim3 blockDim(256);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    const int NSPLIT = 4096;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemm_m8n256k64_v3),   
                hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    // about 76KB for m8n256k64
    // unsigned int dsmem = 2 * (BM * (8 * BK + 8) + BK * (BN + 8)) * sizeof(half);
    unsigned int dsmem = 2 * (BM * (4 * BK + 8) + BK * (BN + 8)) * sizeof(half);
    
    eed_hgemm_m8n256k64_v3<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}


void edgemm_m8n128k64(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 128, BK = 64;
    dim3 blockDim(128);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    const int NSPLIT = 2048;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    // hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemm_m8n256k64_v3),   
    //             hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    // about 39 KB for m8n128k64
    unsigned int dsmem = 2 * (BM * (2 * BK + 8) + BK * (BN + 8)) * sizeof(half);
    
    eed_hgemm_m8n128k64_v4<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}


void edgemm_m8n128k64x4(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 128, BK = 64;
    dim3 blockDim(128);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;
    int BZ = 8;

    dim3 gridDim(BX, BY, BZ);

    // const int NSPLIT = 2048;
    // int split_num = (N + NSPLIT - 1) / NSPLIT;
    // dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    // hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemm_m8n256k64_v3),   
    //             hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    // about 39 KB for m8n128k64
    unsigned int dsmem = 2 * (BM * (2 * BK + 8) + BK * (BN + 8)) * sizeof(half);
    
    eed_hgemm_m8n128k64x4_v7<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}

void edgemm_m8n128k64x4_bt(at::Tensor A, at::Tensor B, at::Tensor C) {
    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 128, BK = 64;
    dim3 blockDim(128);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;
    int BZ = 8;

    dim3 gridDim(BX, BY, BZ);

    // 40.25 KB for m8n128k64
    unsigned int dsmem = 2 * (BM * (2 * BK + 8) + BN * (BK + 8)) * sizeof(half);

    eed_hgemm_m8n128k64x4_v7_bt<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),
        reinterpret_cast<half *>(B.data_ptr<at::Half>()),
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),
        M, N, K);
}

void edgemm_m8n128k128(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 128, BK = 128;
    dim3 blockDim(128);
    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    const int NSPLIT = 2048;
    int split_num = (N + NSPLIT - 1) / NSPLIT;
    dim3 gridDim((BX + split_num - 1) / split_num, BY, split_num);

    hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemm_m8n128k128_v5),   
                hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    // about 74 KB for m8n128k128
    unsigned int dsmem = 2 * (BM * (BK + 8) + BK * (BN + 8)) * sizeof(half);
    
    eed_hgemm_m8n128k128_v5<<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}


void edgemv_m1n128k64x4(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 128, BK = 64;
    dim3 blockDim(128);
    int BX = (N + BN - 1) / BN;
    int BY = 1;
    int BZ = 8;

    dim3 gridDim(BX, BY, BZ);


    // hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemv_m1n128k64_v6),   
    //             hipFuncAttributeMaxDynamicSharedMemorySize, 98304);

    // about 39 KB for m8n128k64
    unsigned int dsmem = (2 * (BM * (2 * BK + 8) + BK * (BN + 8)) + 0) * sizeof(half);
    
    eed_hgemv_m1n128k64x4_v6<8><<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}


void edgemv_m1n256k64x4(at::Tensor A, at::Tensor B, at::Tensor C){

    int M = A.size(0);
    int K = A.size(1);
    int N = B.size(1);

    const int BM = 8, BN = 256, BK = 64;
    dim3 blockDim(256);
    int BX = (N + BN - 1) / BN;
    int BY = 1;
    int BZ = 8;

    dim3 gridDim(BX, BY, BZ);


    hipFuncSetAttribute(reinterpret_cast<const void*>(eed_hgemv_m1n256k64x4_v8<8>),   
                hipFuncAttributeMaxDynamicSharedMemorySize, 76032);

    // about 76 KB for m8n256k64
    unsigned int dsmem = (2 * (BM * (4 * BK + 8) + BK * (BN + 8)) + 0) * sizeof(half);
    
    eed_hgemv_m1n256k64x4_v8<8><<<gridDim, blockDim, dsmem>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        M, N, K
        );
}
