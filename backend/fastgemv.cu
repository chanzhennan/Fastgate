#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>

#include "fastgemv.cuh"

void fastgemv(at::Tensor A, at::Tensor B, at::Tensor C){

    int mat_height_ = A.size(0);
    int vec_height_ = B.size(0);

    int block_dim_x = 128;
    int block_dim_y = 4;
    assert(block_dim_y <= SHARED_MEM_MAX_ROWS);
    assert(block_dim_x * block_dim_y <= MAX_THREADS_PER_BLOCK);
    unsigned int num_per_thread = vec_height_ / block_dim_x;
    assert(num_per_thread >= 8);

    dim3 grid_dim(1, mat_height_ / block_dim_y);
    dim3 block_dim(block_dim_x, block_dim_y);

    gemv_fp16<<<grid_dim, block_dim>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        vec_height_, 
        num_per_thread);
}

void fastgemv_int8(at::Tensor A, at::Tensor B, at::Tensor C){

    int mat_height_ = A.size(0);
    int vec_height_ = B.size(0);

    int block_dim_x = 128;
    int block_dim_y = 4;
    assert(block_dim_y <= SHARED_MEM_MAX_ROWS);
    assert(block_dim_x * block_dim_y <= MAX_THREADS_PER_BLOCK);
    unsigned int num_per_thread = vec_height_ / block_dim_x;
    assert(num_per_thread >= 8);

    dim3 grid_dim(1, mat_height_ / block_dim_y);
    dim3 block_dim(block_dim_x, block_dim_y);

    half zero = __float2half(1.0f);
    half scale = __float2half(0.0f);

    gemv_quantized_int8<<<grid_dim, block_dim>>>(
        reinterpret_cast<int8_t *>(A.data_ptr<int8_t>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        vec_height_, 
        zero, 
        scale,
        num_per_thread);
}

void fastgemv_tuned(at::Tensor A, at::Tensor B, at::Tensor C){

    int mat_height_ = A.size(0);
    int vec_height_ = B.size(0);

    int block_dim_x = 128;
    int block_dim_y = 1;
    assert(block_dim_y <= SHARED_MEM_MAX_ROWS);
    assert(block_dim_x * block_dim_y <= MAX_THREADS_PER_BLOCK);
    unsigned int num_per_thread = vec_height_ / block_dim_x;
    assert(num_per_thread >= 8);

    dim3 grid_dim(1, mat_height_ / block_dim_y);
    dim3 block_dim(block_dim_x, block_dim_y);

    gemv_fp16_tuned<<<grid_dim, block_dim>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        vec_height_, 
        num_per_thread);
}

void fastgemv_extend(at::Tensor A, at::Tensor B, at::Tensor C){

    // A: weight, [N, K]
    // B: vector, [M, K]
    // C: result, [M, N]
    int N = A.size(0);
    int K = B.size(1);
    int M = B.size(0);

    int block_dim_x = 128;
    int block_dim_y = M;
    assert(block_dim_y <= SHARED_MEM_MAX_ROWS);
    assert(block_dim_x * block_dim_y <= MAX_THREADS_PER_BLOCK);
    unsigned int num_per_thread = K / block_dim_x;
    assert(num_per_thread >= 8);

    dim3 grid_dim(N);
    dim3 block_dim(block_dim_x, block_dim_y);

    gemm_fp16<<<grid_dim, block_dim>>>(
        reinterpret_cast<half *>(A.data_ptr<at::Half>()),  
        reinterpret_cast<half *>(B.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(C.data_ptr<at::Half>()),  
        K, N, 
        num_per_thread);
}
