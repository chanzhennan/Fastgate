#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <driver_functions.h>

#include "utility.cuh"

void print_cuda_info() {
  // for fun, just print out some stats on the machine

  int deviceCount = 0;
  hipError_t err = hipGetDeviceCount(&deviceCount);

  printf("---------------------------------------------------------\n");
  printf("Found %d CUDA devices\n", deviceCount);

  for (int i = 0; i < deviceCount; i++) {
    hipDeviceProp_t deviceProps;
    hipGetDeviceProperties(&deviceProps, i);
    printf("Device %d: %s\n", i, deviceProps.name);
    printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
    printf("   Global mem: %.0f MB\n",
           static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
    printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
  }
  printf("---------------------------------------------------------\n");
}

void check(hipError_t result, char const* const func, const char* const file,
           int const line) {
  if (result) {
    fprintf(stderr, "CUDA error = %s at %s:%d '%s'\n",
            hipGetErrorString(result), file, line, func);
    exit(1);
  }
}

__global__ void generate_random_numbers(half* numbers, int Np) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < Np) {
    hiprandState state;
    hiprand_init(clock64(), i, 0, &state);
    numbers[i] = __float2half(hiprand_uniform(&state));
  }
}

__global__ void generate_random_int8_numbers(int8_t* numbers, int Np) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < Np) {
    hiprandState state;
    hiprand_init(clock64(), i, 0, &state);
    numbers[i] = static_cast<int8_t>(hiprand(&state) % 256 -
                                     128);  // Random int8 number [-128, 127]
  }
}

__global__ void generate_random_int4_numbers(uint4_2* numbers, int Np) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < Np) {
    hiprandState state;
    hiprand_init(clock64(), i, 0, &state);
    numbers[i].setX(
        static_cast<uint8_t>(hiprand(&state) % 16));  // Random number [0, 15]
    hiprand_init(clock64(), i, 0, &state);
    numbers[i].setY(
        static_cast<uint8_t>(hiprand(&state) % 16));  // Random number [0, 15]
  }
}